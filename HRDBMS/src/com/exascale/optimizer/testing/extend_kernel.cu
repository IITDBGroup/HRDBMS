
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

extern "C"
{
__constant__ char parseStack[4096];

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int myStrlen(char* string)
{
	char* temp = string;
	while (*temp != 0)
	{
		temp++;
	}
	
	return temp-string;
}

__device__ int parseLong(char* string)
{
	char* temp = string;
	int negative = 0;
	int offset = 0;
	long result = 0;
	int length = myStrlen(string);
		
	if (*temp == '-')
	{
		negative = 1;
		offset = 1;
	}
		
	while (offset < length)
	{
		char b = temp[offset];
		b -= 48;
		result *= 10;
		result += b;
		offset++;
	}
		
	if (negative != 0)
	{
		result *= -1;
	}
		
	return result;
}

__device__ float myStrtod(char* string)
{
	char newTemp[32];
	char* temp = string;
	int p = -1;
	while (*temp != 0 && p == -1)
	{
		if (*temp == '.')
		{
			p = temp - string;
		}
		
		temp++;
	}
	
	temp = string;
	if (p < 0)
	{
		return parseLong(string);
	}
		
	int negative = 0;
	int offset = 0;
	if (*temp == '-')
	{
		negative = 1;
		offset = 1;
	}
	
	int strlen = myStrlen(temp);
	int i = 0;
	while (i < p)
	{
		newTemp[i] = temp[i];
		i++;
	}
	
	i++;
	while (i < strlen)
	{
		newTemp[i-1] = temp[i];
		i++;
	}
	
	temp = newTemp;
	long n = parseLong(temp);
	int x = strlen - p - offset;
	i = 0;
	long d = 1;
	while (i < x)
	{
		d *= 10;
		i++;
	}
		
	float retval = (n*1.0f) / (d*1.0f);
	if (negative != 0)
	{
		retval *= -1.0f;
	}
	
	return retval;
}

__global__ void doExtendKernel(float* deviceRows, float* deviceResults, int numJobs, int numCols, int numPrefixes, int prefixBytesLength, float* execStack)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numJobs)
    {
    	int parseStackPtr = 0;
		int parseStackProcessed = 0;
		int esp = 512 * idx;
		int rowsCntr = 0;

		while (parseStackProcessed < numPrefixes)
		{ 
			char* temp = parseStack + parseStackPtr;
			if (*temp == '*') 
			{
				esp--;
				float lhs = execStack[esp];
				esp--;
				float rhs = execStack[esp];
				execStack[esp] = lhs * rhs;
				esp++;
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else if (*temp == '-') 
			{
				esp--;
				float lhs = execStack[esp];
				esp--;
				float rhs = execStack[esp];
				execStack[esp] = lhs - rhs;
				esp++;
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else if (*temp == '+') 
			{
				esp--;
				float lhs = execStack[esp];
				esp--;
				float rhs = execStack[esp];
				execStack[esp] = lhs + rhs;
				esp++;
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else if (*temp == '/') 
			{
				esp--;
				float lhs = execStack[esp];
				esp--;
				float rhs = execStack[esp];
				execStack[esp] = lhs / rhs;
				esp++;
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else 
			{
				if ((*temp >= 'a' && *temp <= 'z') || (*temp >= 'A' && *temp <= 'Z') || (*temp == '_')) 
				{
					execStack[esp] = deviceRows[rowsCntr + idx * numCols];
					rowsCntr++;
					esp++;
					parseStackPtr += (1 + myStrlen(temp));
					parseStackProcessed++;
				} 
				else 
				{
					float d = myStrtod(temp);
					execStack[esp] = d;
					esp++;
					parseStackPtr += (1 + myStrlen(temp));
					parseStackProcessed++;
				}
			}
		}

		esp--;
		deviceResults[idx] = execStack[esp];
	}
}

void cudaExtend(float* nativeRows, char* nativePrefix, float* nativeResults, int numJobs, int numCols, int numPrefixes, int prefixBytesLength)
{
	float* deviceResults;
	float* deviceRows;
	//cuda malloc deviceResults
	gpuErrchk(hipMalloc((void**)&deviceResults, numJobs * sizeof(float))); 
	//gpuErrchk(cudaMemset((void*)deviceResults, 0xFE, numJobs * sizeof(float)));
	//cuda memcpy prefix
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(parseStack), nativePrefix, prefixBytesLength));
	//cuda malloc rows
	gpuErrchk(hipMalloc((void**)&deviceRows, sizeof(float) * numJobs * numCols));
	//cuda memcpy rows
	gpuErrchk(hipMemcpy(deviceRows, nativeRows, sizeof(float) * numJobs * numCols, hipMemcpyHostToDevice));
	float* execStack;
	gpuErrchk(hipMalloc((void**)&execStack, sizeof(float) * 512 * numJobs));
	//invoke kernel
	int blockSize = 128;
	int nBlocks = numJobs/blockSize + (numJobs%blockSize == 0?0:1);
	doExtendKernel <<< nBlocks, blockSize >>> (deviceRows, deviceResults, numJobs, numCols, numPrefixes, prefixBytesLength, execStack);
	//copy deviceResults back to nativeResults
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipMemcpy(nativeResults, deviceResults, numJobs * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(deviceRows));
	gpuErrchk(hipFree(deviceResults));
	gpuErrchk(hipFree(execStack));
}
}