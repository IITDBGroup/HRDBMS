
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

__device__ int myStrlen(char* string)
{
	int cnt = 0;
	char* temp = string;
	while (*temp != 0)
	{
		cnt++;
		temp++;
	}
	
	return cnt;
}

__device__ int parseLong(char* string)
{
	char* temp = string;
	int negative = 0;
	int offset = 0;
	long result = 0;
	int length = myStrlen(string);
		
	if (*temp == '-')
	{
		negative = 1;
		offset = 1;
	}
		
	while (offset < length)
	{
		char b = temp[offset];
		b -= 48;
		result *= 10;
		result += b;
		offset++;
	}
		
	if (negative != 0)
	{
		result *= -1;
	}
		
	return result;
}

__device__ double myStrtod(char* string)
{
	char newTemp[32];
	char* temp = string;
	int p = -1;
	while (*temp != 0 && p == -1)
	{
		if (*temp == '.')
		{
			p = temp - string;
		}
		
		temp++;
	}
	
	temp = string;
	if (p < 0)
	{
		return parseLong(string);
	}
		
	int negative = 0;
	int offset = 0;
	if (*temp == '-')
	{
		negative = 1;
		offset = 1;
	}
	
	int strlen = myStrlen(temp);
	int i = 0;
	while (i < p)
	{
		newTemp[i] = temp[i];
		i++;
	}
	
	i++;
	while (i < strlen)
	{
		newTemp[i-1] = temp[i];
		i++;
	}
	
	temp = newTemp;
	long n = parseLong(temp);
	int x = strlen - p - offset;
	i = 0;
	long d = 1;
	while (i < x)
	{
		d *= 10;
		i++;
	}
		
	double retval = (n*1.0) / (d*1.0);
	if (negative != 0)
	{
		retval *= -1;
	}
	
	return retval;
}

__global__ void doExtendKernel(double* deviceRows, char* parseStack, double* deviceResults, int numJobs, int numCols, int numPrefixes, int prefixBytesLength)
{
	char execStack[4096];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numJobs)
    {
		int parseStackPtr = 0;
		int parseStackProcessed = 0;
		int execStackPtr = 0;
		int rowsCntr = 0;

		while (parseStackProcessed < numPrefixes)
		{ 
			char* temp = parseStack + parseStackPtr;
			if (*temp == '*') 
			{
				execStackPtr -= sizeof(double);
				double lhs = *(double*)(execStack + execStackPtr);
				execStackPtr -= sizeof(double);
				double rhs = *(double*)(execStack + execStackPtr);
				*(double*)(execStack + execStackPtr) = lhs * rhs;
				execStackPtr += sizeof(double);
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else if (*temp == '-') 
			{
				execStackPtr -= sizeof(double);
				double lhs = *(double*)(execStack + execStackPtr);
				execStackPtr -= sizeof(double);
				double rhs = *(double*)(execStack + execStackPtr);
				*(double*)(execStack + execStackPtr) = lhs - rhs;
				execStackPtr += sizeof(double);
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else if (*temp == '+') 
			{
				execStackPtr -= sizeof(double);
				double lhs = *(double*)(execStack + execStackPtr);
				execStackPtr -= sizeof(double);
				double rhs = *(double*)(execStack + execStackPtr);
				*(double*)(execStack + execStackPtr) = lhs + rhs;
				execStackPtr += sizeof(double);
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else if (*temp == '/') 
			{
				execStackPtr -= sizeof(double);
				double lhs = *(double*)(execStack + execStackPtr);
				execStackPtr -= sizeof(double);
				double rhs = *(double*)(execStack + execStackPtr);
				*(double*)(execStack + execStackPtr) = lhs / rhs;
				execStackPtr += sizeof(double);
				parseStackPtr += 2;
				parseStackProcessed += 1;
			} 
			else 
			{
				if ((*temp >= 'a' && *temp <= 'z') || (*temp >= 'A' && *temp <= 'Z') || (*temp == '_')) 
				{
					*(double*)(execStack + execStackPtr) = deviceRows[rowsCntr + idx * numCols];
					rowsCntr++;
					execStackPtr += sizeof(double);
					parseStackPtr += (1 + myStrlen(temp));
					parseStackProcessed++;
				} 
				else 
				{
					double d = myStrtod(temp);
					*(double*)(execStack + execStackPtr) = d;
					execStackPtr += sizeof(double);
					parseStackPtr += (1 + myStrlen(temp));
					parseStackProcessed++;
				}
			}
		}

		execStackPtr -= sizeof(double);
		deviceResults[idx] = *(double*)(execStack + execStackPtr);
	}
}

void cudaExtend(double* nativeRows, char* nativePrefix, double* nativeResults, int numJobs, int numCols, int numPrefixes, int prefixBytesLength)
{
	double* deviceResults;
	char* devicePrefix;
	double* deviceRows;
	//cuda malloc deviceResults
	hipMalloc((void**)&deviceResults, numJobs * sizeof(double)); 
	//cuda malloc prefix
	hipMalloc((void**)&devicePrefix, prefixBytesLength);
	//cuda memcpy prefix
	hipMemcpy(devicePrefix, nativePrefix, prefixBytesLength, hipMemcpyHostToDevice);
	//cuda malloc rows
	hipMalloc((void**)&deviceRows, sizeof(double) * numJobs * numCols);
	//cuda memcpy rows
	hipMemcpy(deviceRows, nativeRows, sizeof(double) * numJobs * numCols, hipMemcpyHostToDevice);
	//invoke kernel
	int blockSize = 128;
	int nBlocks = numJobs/blockSize + (numJobs%blockSize == 0?0:1);
	doExtendKernel <<< nBlocks, blockSize >>> (deviceRows, devicePrefix, deviceResults, numJobs, numCols, numPrefixes, prefixBytesLength);
	//copy deviceResults back to nativeResults
	hipMemcpy(nativeResults, deviceResults, numJobs * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(deviceRows);
	hipFree(devicePrefix);
	hipFree(deviceResults);
}
